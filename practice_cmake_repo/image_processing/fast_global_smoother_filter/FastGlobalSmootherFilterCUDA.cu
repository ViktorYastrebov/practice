#include "hip/hip_runtime.h"
#include "FastGlobalSmootherFilterCUDA.h"
#include "hip/hip_runtime.h"
#include <opencv2/core/cuda_stream_accessor.hpp>

namespace impl {

    //INFO: original
    //inline float get_negative_weight(float p1, float p2, bool zero_norm) {
    //    //Farbman's weight works better than the one involving exp.
    //    if (zero_norm)
    //        return -1.0f / ((p1 - p2)*(p1 - p2) + 1e-7f);
    //    return -1.0f / (abs(p1 - p2) + 1e-4f);
    //}

    template<bool isZeroNormalization = false>
    struct NegativeWeightProcessor {
        float __device__ operator()(const float p1, const float p2) {
            return -1.0f / (fabsf(p1 - p2) + 1e-4f);
        }
    };

    template<>
    struct  NegativeWeightProcessor<true> {
        float __device__ operator()(const float p1, const float p2) {
            return -1.0f / ((p1 - p2)*(p1 - p2) + 1e-7f);
        }
    };

    __global__ void processHorizontalWeightsKernel(const cv::cuda::PtrStepSzf input, cv::cuda::PtrStepSzf output)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        int nextX = x + 1;

        bool xValueInBound = x >= 0 && x <= input.cols - 2;
        bool nextXValueInBound = nextX >= 1 && nextX <= input.cols - 1;
        bool yValueInBound = y >= 0 && y <= input.rows - 1;

        if (xValueInBound && nextXValueInBound && yValueInBound) {
            //output(y, x) = NegativeWeightProcessor<true>()(input(y, x), input(y, nextX));
             output(y, x) = NegativeWeightProcessor<true>()(input(y, x), input(y, nextX));
        }

        bool isLastX = (x == (input.cols - 1));

        if (isLastX && yValueInBound) {
            output(y, x) = 0;
        }
    }

    __global__ void processVerticalWeightsKernel(const cv::cuda::PtrStepSzf input, cv::cuda::PtrStepSzf output) {

        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        int nextY = y + 1;

        bool xValueInBound = x >= 0 && x <= input.cols - 1;
        bool yValueInBound = y >= 0 && y <= input.rows - 2;
        bool nextYValueInBound = nextY >= 1 && nextY <= input.rows - 1;

        if (xValueInBound && yValueInBound && nextYValueInBound) {
            output(y, x) = NegativeWeightProcessor<true>()(input(y, x), input(nextY, x));
        }
        
        bool isLastY = (y == (input.rows - 1));
        if (isLastY && xValueInBound) {
            output(y, x) = 0;
        }
    }

    __global__ void process4RowsPerBlock(cv::cuda::PtrStepSzf chorData, cv::cuda::PtrStepSzf interD, cv::cuda::PtrStepSzf current, float lambda, int width) {
        int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;

        if (rowIdx >= 0 && rowIdx <= current.rows - 1) {
            cv::cuda::PtrStepSzf::elem_type *chorRowPtr = chorData.ptr(rowIdx);
            cv::cuda::PtrStepSzf::elem_type *interRowPtr = interD.ptr(rowIdx);
            cv::cuda::PtrStepSzf::elem_type *currentRowPtr = current.ptr(rowIdx);

            float coefPrev = lambda * chorRowPtr[0];

            //INFO: head processing(forward pass) for X=0 elements
            interRowPtr[0] = coefPrev / (1 - coefPrev);
            currentRowPtr[0] = currentRowPtr[0] / (1 - coefPrev);

            //TODO: think of removing True data dependecies => might be parallelized on grid !!!
            for (int i = 1; i < width; ++i) {

                coefPrev = lambda * chorRowPtr[i - 1];
                float coefCur = lambda * chorRowPtr[i];
                float denom = (1 - coefPrev - coefCur) - interRowPtr[i - 1] * coefPrev;
                interRowPtr[i] = coefCur / denom;
                currentRowPtr[i] = (currentRowPtr[i] - currentRowPtr[i - 1] * coefPrev) / denom;
            }

            //TODO: think of removing True data dependecies => might be parallelized on grid !!!
            //backward pass:
            for (int i = width - 2; i >= 0; --i) {
                chorRowPtr[i] = chorRowPtr[i] - interRowPtr[i] * chorRowPtr[i + 1];
            }
        }
    }

    __global__ void processRow(cv::cuda::PtrStepSzf chorData, cv::cuda::PtrStepSzf interD, cv::cuda::PtrStepSzf current, float lambda, int width) {
        int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;

        if (rowIdx >= 0 && rowIdx <= current.rows - 1) {
            cv::cuda::PtrStepSzf::elem_type *chorRowPtr = chorData.ptr(rowIdx);
            cv::cuda::PtrStepSzf::elem_type *interRowPtr = interD.ptr(rowIdx);
            cv::cuda::PtrStepSzf::elem_type *currentRowPtr = current.ptr(rowIdx);

            //forward pass:
            float coefPrev = lambda * chorRowPtr[0];
            interRowPtr[0] = coefPrev / (1 - coefPrev);
            currentRowPtr[0] = currentRowPtr[0] / (1 - coefPrev);

            for (int i = 1; i < width; ++i) {
                float coefCur = lambda * chorRowPtr[i];
                float denom = (1 - coefPrev - coefCur) - interRowPtr[i - 1] * coefPrev;

                interRowPtr[i] = coefCur / denom;
                currentRowPtr[i] = (currentRowPtr[i] - currentRowPtr[i - 1] * coefPrev) / denom;
                coefPrev = coefCur;
            }

            //backward pass:
            for (int i = width - 2; i >= 0; i--) {
                currentRowPtr[i] = currentRowPtr[i] - interRowPtr[i] * currentRowPtr[i + 1];
            }
        }
    }

    __global__ void processVerticaForwardlPassFirstRow(cv::cuda::PtrStepSzf cVertFirstRow, cv::cuda::PtrStepSzf interDFirstRow, cv::cuda::PtrStepSzf currentFirstRow, float lambda) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        constexpr const int FIRST_ROW_IDX = 0;

        if (idx >= 0 && idx < currentFirstRow.rows) {
            float coefCurrent = lambda * cVertFirstRow.ptr(FIRST_ROW_IDX)[idx];
            interDFirstRow.ptr(FIRST_ROW_IDX)[idx] = coefCurrent / (1 - coefCurrent);
            currentFirstRow.ptr(FIRST_ROW_IDX)[idx] = currentFirstRow.ptr(FIRST_ROW_IDX)[idx] / (1 - coefCurrent);
        }
    }
    //INFO: it's not possible to get the cv::cuda::GpuMat row ptr outside the kernek, so let's pass just the IDXs
    // Generaly this algo has 2 loop and outer one has the TRUE data dependecies. So just parallize by inner one

    // INFO:
    //      coef_prev = fgs->lambda*Cvert_row_prev[j];
    //      coef_cur = fgs->lambda*Cvert_row[j];
    //      denom = (1 - coef_prev - coef_cur) - interD_row_prev[j] * coef_prev;
    //      interD_row[j] = coef_cur / denom;
    //      cur_row[j] = (cur_row[j] - cur_row_prev[j] * coef_prev) / denom;

    __global__ void processVerticalForwardPass(cv::cuda::PtrStepSzf cVertMat, cv::cuda::PtrStepSzf interD, cv::cuda::PtrStepSzf current, float lambda, const int currentRowIdx, const int prevRowIdx) {
        int jIdx = blockIdx.x * blockDim.x + threadIdx.x;
        if (jIdx > 0 && jIdx < current.cols) {
            float coefPrev = lambda * cVertMat.ptr(prevRowIdx)[jIdx];
            float coefCurrent = lambda * cVertMat.ptr(currentRowIdx)[jIdx];
            float denom = (1 - coefPrev - coefCurrent) - interD.ptr(prevRowIdx)[jIdx] * coefPrev;
            interD.ptr(currentRowIdx)[jIdx] = coefCurrent / denom;
            current.ptr(currentRowIdx)[jIdx] = (current.ptr(currentRowIdx)[jIdx] - current.ptr(prevRowIdx)[jIdx] * coefPrev) / denom;
        }
    }

    //INFO:
    //  backward pass:
    //  for (int i = fgs->h - 2; i >= 0; i--)
    //  {
    //    interD_row = (WorkType*)fgs->interD.ptr(i);
    //    cur_row = (WorkType*)cur->ptr(i);
    //    cur_row_next = (WorkType*)cur->ptr(i + 1);
    //    int j = start;
    //    //USE CUDA FOR INNER LOOP
    //    for (; j < end; j++)
    //        cur_row[j] = cur_row[j] - interD_row[j] * cur_row_next[j];
    //  }

    __global__ void processVerticalBackwardPass(cv::cuda::PtrStepSzf interD, cv::cuda::PtrStepSzf current, const int currentRowIdx, const int nextRowIdx) {
        int jIdx = blockIdx.x * blockDim.x + threadIdx.x;
        if (jIdx > 0 && jIdx < current.cols) {
            current.ptr(currentRowIdx)[jIdx] = current.ptr(currentRowIdx)[jIdx] - interD.ptr(currentRowIdx)[jIdx] * current.ptr(nextRowIdx)[jIdx];
        }
    }
}

namespace ni {
    namespace cuda {
        namespace algorithms {

            void processHorizontalWeightWrapper(const cv::cuda::GpuMat &input, cv::cuda::GpuMat &output, cv::cuda::Stream &stream) {
                dim3 cthreads(16, 16);
                dim3 cblocks(
                    static_cast<int>(std::ceil(input.size().width /
                        static_cast<double>(cthreads.x))),
                    static_cast<int>(std::ceil(input.size().height /
                        static_cast<double>(cthreads.y))));

                hipStream_t cudaStream = cv::cuda::StreamAccessor::getStream(stream);
                impl::processHorizontalWeightsKernel << < cblocks, cthreads, 0, cudaStream >> > (input, output);

                hipError_t error = hipGetLastError();
                if (error != hipSuccess) {
                    throw std::runtime_error(hipGetErrorString(error));
                }
            }

            void processVerticalWeightWrapper(const cv::cuda::GpuMat &input, cv::cuda::GpuMat &output, cv::cuda::Stream &stream) {
                dim3 cthreads(16, 16);
                dim3 cblocks(
                    static_cast<int>(std::ceil(input.size().width /
                        static_cast<double>(cthreads.x))),
                    static_cast<int>(std::ceil(input.size().height /
                        static_cast<double>(cthreads.y))));

                hipStream_t cudaStream = cv::cuda::StreamAccessor::getStream(stream);
                impl::processVerticalWeightsKernel << < cblocks, cthreads, 0, cudaStream >> > (input, output);

                hipError_t error = hipGetLastError();
                if (error != hipSuccess) {
                    throw std::runtime_error(hipGetErrorString(error));
                }
            }

            void horizontalPass(cv::cuda::GpuMat &chorMat, cv::cuda::GpuMat &interDMat, cv::cuda::GpuMat &current, cv::cuda::Stream &stream, float lambda) {

                if (chorMat.size() != current.size() || interDMat.size() != current.size()) {
                    throw std::logic_error("Invalid matrix size");
                }

                //total processors = Height
                const int BLOCKS = 16;
                const int THREADS_PER_BLOCK = static_cast<int>(std::ceil(current.size().height / static_cast<double>(BLOCKS) ));

                hipStream_t cudaStream = cv::cuda::StreamAccessor::getStream(stream);
                impl::process4RowsPerBlock << < BLOCKS, THREADS_PER_BLOCK, 0, cudaStream >> > (chorMat, interDMat, current, lambda, current.size().width);
                hipError_t error = hipGetLastError();
                if (error != hipSuccess) {
                    throw std::runtime_error(hipGetErrorString(error));
                }

                impl::processRow << < BLOCKS, THREADS_PER_BLOCK, 0, cudaStream >> > (chorMat, interDMat, current, lambda, current.size().width);
                error = hipGetLastError();
                if (error != hipSuccess) {
                    throw std::runtime_error(hipGetErrorString(error));
                }
            }

            //INFO: original algo also has the outer loop true data dependecies
            //      so it can be parallized by inner loop only
            void verticalPass(cv::cuda::GpuMat &cVertMat, cv::cuda::GpuMat &interDMat, cv::cuda::GpuMat &current, cv::cuda::Stream &stream, float lambda, const int height) {
                if (cVertMat.size() != current.size() || interDMat.size() != current.size()) {
                    throw std::logic_error("Invalid matrix size");
                }

                const int BLOCKS = 16;
                const int THREADS_PER_BLOCK = static_cast<int>(std::ceil(current.size().width / static_cast<double>(BLOCKS)));
                hipStream_t cudaStream = cv::cuda::StreamAccessor::getStream(stream);

                impl::processVerticaForwardlPassFirstRow << < BLOCKS, THREADS_PER_BLOCK, 0, cudaStream >> > (cVertMat, interDMat, current, lambda);

                for (int i = 1; i < height; ++i) {
                    impl::processVerticalForwardPass << < BLOCKS, THREADS_PER_BLOCK, 0, cudaStream >> > (cVertMat, interDMat, current, lambda, i, i - 1);
                }

                for (int i = height - 2; i >= 0; i--) {
                    impl::processVerticalBackwardPass << < BLOCKS, THREADS_PER_BLOCK, 0, cudaStream >> > (interDMat, current, i, i + 1);
                }

            }
        }
    }
}
